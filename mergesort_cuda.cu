#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cutil_inline.h>

// size of list
#define NUM    64

__device__ inline
void merge(int* arr, int* new_arr, int left, int right, int upper_bound) {
  int i = left;
  int j = right;
  int k = left;

  while (i < right && j < upper_bound) { 
    if (arr[i] <= arr[j]) {
      new_arr[k] = arr[i];
      i++;
    } 
    else {
      new_arr[k] = arr[j];
      j++;
    }
    k++;
  }
  
  while (i < right) { 
    new_arr[k] = arr[i];
    i++;
    k++;
  }
  
  while (j < upper_bound) { 
    new_arr[k] = arr[j];
    j++;
    k++;
  }

  for (k = left; k < upper_bound; k++) { 
    arr[k] = new_arr[k]; 
  }
}

__global__ static void merge_sort(int* values, int* results) {
  
  extern __shared__ int shared[];
  const unsigned int tid = threadIdx.x;
  
  int k;
  int upper_bound;
  int i;

  shared[tid] = values[tid]; // input -> shared memory
  
  __syncthreads();
  
  k = 1;
  while(k < NUM) {
    i = 1;
    while(i + k <= NUM) {
        upper_bound = i + k * 2;
        if (upper_bound > NUM) {
          upper_bound = NUM + 1;
        }
        merge(shared, results, i, i + k, upper_bound);
        i = i + k * 2;
    }
    k = k * 2;
    __syncthreads();
  }
  
  values[tid] = shared[tid];
}

int main(int argc, char** argv) {
	if (cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
		cutilDeviceInit(argc, argv);
  }
	else {
		hipSetDevice( cutGetMaxGflopsDeviceId() );
  }

  int values[NUM];

  for (int i = 0; i < NUM; i++) {
    values[i] = rand();
  }

  int* dvalues;
  int* results;
  cutilSafeCall(hipMalloc((void**)&dvalues, sizeof(int) * NUM));
  cutilSafeCall(hipMemcpy(dvalues, values, sizeof(int) * NUM, hipMemcpyHostToDevice));
  cutilSafeCall(hipMalloc((void**)&results, sizeof(int) * NUM));
  cutilSafeCall(hipMemcpy(results, values, sizeof(int)* NUM, hipMemcpyHostToDevice));

  merge_sort<<<1, NUM, sizeof(int) * NUM*2>>>(dvalues, results);

  cutilCheckMsg("Kernel execution failed");

  bool passed = true;
  for(int i = 1; i < NUM; i++) {
    if (values[i-1] > values[i]) {
      passed = false;
    }
  }
  printf( "Test %s\n", passed ? "PASSED" : "FAILED");

  cutilSafeCall(hipFree(dvalues));
  cutilSafeCall(hipMemcpy(values, results, sizeof(int) * NUM, hipMemcpyDeviceToHost));
  cutilSafeCall(hipFree(results));

  hipDeviceReset();
  cutilExit(argc, argv);

  int int_size = sizeof(int);

  adiak::init(NULL);
  adiak::launchdate();    // launch date of the job
  adiak::libraries();     // Libraries used
  adiak::cmdline();       // Command line used to launch the job
  adiak::clustername();   // Name of the cluster
  adiak::value("Algorithm", "MergeSort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
  adiak::value("ProgrammingModel", "CUDA"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
  adiak::value("Datatype", "int"); // The datatype of input elements (e.g., double, int, float)
  adiak::value("SizeOfDatatype", int_size); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
  adiak::value("InputSize", NUM); // The number of elements in input dataset (1000)
  adiak::value("InputType", Random); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
  adiak::value("num_procs", num_procs); // The number of processors (MPI ranks)
  adiak::value("num_threads", num_threads); // The number of CUDA or OpenMP threads
  adiak::value("num_blocks", num_blocks); // The number of CUDA blocks 
  adiak::value("group_num", 17); // The number of your group (integer, e.g., 1, 10)
  adiak::value("implementation_source", "Online + Handwritten") // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").

}
