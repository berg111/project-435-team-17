#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <vector>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>



#define BLOCK_SIZE 256

// Function to swap two elements
__device__ void swap(int *a, int *b) {
    int temp = *a;
    *a = *b;
    *b = temp;
}

// Function to perform partition
__device__ int partition(int *arr, int low, int high) {
    int pivot = arr[high];
    int i = (low - 1);

    for (int j = low; j <= high - 1; j++) {
        if (arr[j] < pivot) {
            i++;
            swap(&arr[i], &arr[j]);
        }
    }
    swap(&arr[i + 1], &arr[high]);
    return (i + 1);
}

// Function for iterative Quicksort using a stack
__global__ void quicksort(int *arr, int low, int high) {
    int stack[32]; // Arbitrary size for the stack
    int top = -1;

    stack[++top] = low;
    stack[++top] = high;

    while (top >= 0) {
        high = stack[top--];
        low = stack[top--];

        int pi = partition(arr, low, high);

        if (pi - 1 > low) {
            stack[++top] = low;
            stack[++top] = pi - 1;
        }

        if (pi + 1 < high) {
            stack[++top] = pi + 1;
            stack[++top] = high;
        }
    }
}




//Helper function to create array of random values
void generate_array(int* array, int size){
    
    // array.resize(size);

	for(int i = 0; i < size; i++){
		array[i] = rand() % 100000;
    }
}

bool isSorted(int* array, int array_size){
    for(int i = 0; i < array_size - 1; ++i){
        if(array[i] > array[i + 1]){
            return false;
        }
    }
    return true;
}


int main(int argc, char** argv){

    CALI_CXX_MARK_FUNCTION;

    int threads = atoi(argv[1]);
    int array_size = atoi(argv[2]);

    cali::ConfigManager mgr;
    mgr.start();

    clock_t start, stop;

    int array[array_size] = {0};

    generate_array(array, array_size);
    for (int i = 0; i < array_size; i++) {
        std::cout << array[i] << ", ";
    }
    std::cout << std::endl;
    std::cout << array_size << std::endl;

    int* gpu_array;
    size_t size = array_size * sizeof(int);
    hipError_t cudaStatus;
    cudaStatus = hipMalloc((void**) &gpu_array, size);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }
    cudaStatus = hipMemcpy(gpu_array, array, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy HostToDevice failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(gpu_array);
        return 1;
    }
    start = clock();

    quicksort<<<1, 1>>>(gpu_array, 0, array_size - 1);
    // hipDeviceSynchronize();
    stop = clock();

    double elapsed = (double)(stop - start) / CLOCKS_PER_SEC;




    cudaStatus = hipMemcpy(array, gpu_array, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy DeviceToHost failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(gpu_array);
        return 1;
    }

    if (isSorted(array, array_size)) {
        std::cout << "Array is sorted." << std::endl;
    } else {
        std::cout << "Array is not sorted." << std::endl;
    }
    for (int i = 0; i < array_size; i++) {
        std::cout << array[i] << ", ";
    }

    hipFree(gpu_array);

    adiak::init(NULL);
    adiak::user();
    adiak::launchdate();
    adiak::libraries();
    adiak::cmdline();
    adiak::clustername();
    adiak::value("num_threads", threads);
    adiak::value("num_vals", array_size);
    adiak::value("Sort_time", elapsed);

    mgr.stop();
    mgr.flush();


}