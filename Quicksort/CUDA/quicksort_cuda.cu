#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <vector>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>



#define BLOCK_SIZE 256
float timeA, timeB, timeC, bandwidth = 0.0;

// Function to swap two elements
__device__ void swap(int *a, int *b) {
    int temp = *a;
    *a = *b;
    *b = temp;
}

// Function to perform partition
__device__ int partition(int *arr, int low, int high) {
    int pivot = arr[high];
    int i = (low - 1);

    for (int j = low; j <= high - 1; j++) {
        if (arr[j] < pivot) {
            i++;
            swap(&arr[i], &arr[j]);
        }
    }
    swap(&arr[i + 1], &arr[high]);
    return (i + 1);
}

// Function for iterative Quicksort using a stack
__global__ void quicksort(int *arr, int low, int high) {
    int stack[32]; // Arbitrary size for the stack
    int top = -1;

    stack[++top] = low;
    stack[++top] = high;

    while (top >= 0) {
        high = stack[top--];
        low = stack[top--];

        int pi = partition(arr, low, high);

        if (pi - 1 > low) {
            stack[++top] = low;
            stack[++top] = pi - 1;
        }

        if (pi + 1 < high) {
            stack[++top] = pi + 1;
            stack[++top] = high;
        }
    }
}




//Helper function to create array of random values
void generate_array(int* array, int size){
    
    // array.resize(size);

	for(int i = 0; i < size; i++){
		array[i] = i;
    }
}

bool isSorted(int* array, int array_size){
    for(int i = 0; i < array_size - 1; ++i){
        if(array[i] > array[i + 1]){
            return false;
        }
    }
    return true;
}


int main(int argc, char** argv){

    CALI_CXX_MARK_FUNCTION;

    hipEvent_t h_t_d_start, h_t_d_end, sort_step_start, sort_step_end, d_t_h_start, d_t_h_end;

    int threads = atoi(argv[1]);
    int array_size = atoi(argv[2]);

    cali::ConfigManager mgr;
    mgr.start();

    // clock_t start, stop;

    int array[array_size] = {0};

    generate_array(array, array_size);
    // for (int i = 0; i < array_size; i++) {
    //     std::cout << array[i] << ", ";
    // }
    std::cout << std::endl;
    std::cout << array_size << std::endl;

    int* gpu_array;
    size_t size = array_size * sizeof(int);
    hipError_t cudaStatus;
    cudaStatus = hipMalloc((void**) &gpu_array, size);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }
    cudaStatus = hipMemcpy(gpu_array, array, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy HostToDevice failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(gpu_array);
        return 1;
    }
    // start = clock();

    CALI_MARK_BEGIN("comp");
    hipEventCreate(&sort_step_start);
    hipEventCreate(&sort_step_end);
    hipEventRecord(sort_step_start);

    quicksort<<<1, threads>>>(gpu_array, 0, array_size - 1);
    hipDeviceSynchronize();


    hipEventRecord(sort_step_end);
    hipEventSynchronize(sort_step_end);
    hipEventElapsedTime(&timeB, sort_step_start, sort_step_end);

    CALI_MARK_END("comp");
    // stop = clock();

    // double elapsed = (double)(stop - start) / CLOCKS_PER_SEC;




    cudaStatus = hipMemcpy(array, gpu_array, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy DeviceToHost failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(gpu_array);
        return 1;
    }

    printf("Time elapsed: %f ", timeB / 1000);

    if (isSorted(array, array_size)) {
        std::cout << "Array is sorted." << std::endl;
    } else {
        std::cout << "Array is not sorted." << std::endl;
    }
    // for (int i = 0; i < array_size; i++) {
    //     std::cout << array[i] << ", ";
    // }

    hipFree(gpu_array);

    // adiak::init(NULL);
    // adiak::user();
    // adiak::launchdate();
    // adiak::libraries();
    // adiak::cmdline();
    // adiak::clustername();
    // adiak::value("num_threads", threads);
    // adiak::value("num_vals", array_size);
    // // adiak::value("Sort_time", elapsed);

            adiak::init(NULL);
        adiak::launchdate();    // launch date of the job
        adiak::libraries();     // Libraries used
        adiak::cmdline();       // Command line used to launch the job
        adiak::clustername();   // Name of the cluster
        adiak::value("Algorithm", "QuickSort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
        adiak::value("ProgrammingModel", "CUDA"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
        adiak::value("Datatype", "float"); // The datatype of input elements (e.g., double, int, float)
        adiak::value("SizeOfDatatype", sizeof(float)); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
        adiak::value("InputSize", array_size); // The number of elements in input dataset (1000)
        adiak::value("InputType", "Random"); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
        adiak::value("num_threads", threads); // The number of processors (MPI ranks)
        // adiak::value("num_threads", num_threads); // The number of CUDA or OpenMP threads
        // adiak::value("num_blocks", num_blocks); // The number of CUDA blocks 
        adiak::value("group_num", 17); // The number of your group (integer, e.g., 1, 10)
        adiak::value("implementation_source", "Online"); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").

    mgr.stop();
    mgr.flush();


}